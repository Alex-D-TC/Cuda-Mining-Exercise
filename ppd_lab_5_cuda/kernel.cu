#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <random>
#include <stdio.h>
#include <string>
#include <iostream>
#include <time.h>

#define ROR(bytes, cnt) ( (bytes >> cnt) | ( bytes << ((sizeof(bytes) * 8) - cnt) ) )
#define ROL(bytes, cnt) ( (bytes << cnt) | ( bytes >> ((sizeof(bytes) * 8) - cnt) ) )
#define CUDA_HANDLE_ERR(err, desc_str) if(err != hipSuccess) printf("%s: %s\n", desc_str, hipGetErrorString(err));
#define CUDA_HANDLE_ERR_BOOL_FLAG(err, desc_str, flag) if(err != hipSuccess) { printf("%s: %s\n", desc_str, hipGetErrorString(err)); flag = true; }

__host__ __device__ bool increment(uint8_t *bytes, uint32_t bytes_len) {
	bool overflow = true;
	for (int64_t i = bytes_len - 1; overflow && i >= 0; --i) {
		bytes[i] += 1;
		overflow = bytes[i] == 0;
	}
	return overflow;
}

__host__ __device__ uint32_t zero_count(uint8_t *in_b, size_t in_b_len) {

	uint32_t count = 0;
	bool done = false;
	for (size_t i = 0; !done && i < in_b_len; ++i) {
		uint8_t data = in_b[i];
		for (char j = 0; !done && j < 8; ++j) {
			if (((data & (1 << 7)) >> 7) % 2 == 0)
				count += 1;
			else
				done = true;
			data <<= 1;
		}
	}

	return count;
}

__host__ __device__ void hash_256(const uint8_t *in_b, size_t in_b_len, uint8_t *out_b) {
	uint32_t a = 0x6a09e667, b = 0xbb67ae85, c = 0x3c6ef372, d = 0xa54ff53a,
		e = 0x510e527f, f = 0x9b05688c, g = 0x1f83d9ab, h = 0x5be0cd19;

	// prepare padding (if needed)
	uint64_t to_pad = 32 - in_b_len % 32;
	uint64_t block_cnt = (in_b_len + to_pad) / 32;
	
	uint32_t h0 = a, h1 = b, h2 = c, h3 = d, h4 = e, h5 = f, h6 = g, h7 = h;

	// compute
	for (size_t run = 0; run < block_cnt; ++run) {

		a = h0;
		b = h1;
		c = h2;
		d = h3;
		e = h4;
		f = h5;
		g = h6;
		h = h7;

		size_t step = 0;
		for (size_t block_idx = run * 32; step < 8; ++step, block_idx += 4) {
			uint32_t block = 0;
			uint8_t block_size = sizeof(block);
			size_t overflow_len = 0;

			if (block_idx + 4 > in_b_len)
				overflow_len = block_idx + block_size - in_b_len;

			if (overflow_len > 4)
				overflow_len = 4;

			if (block_idx < in_b_len) {
				/*for (size_t i = 0; i < block_size - overflow_len; ++i) {
					*((uint8_t*)(&block) + i) = *(in_b + block_idx + i);
				}*/
				std::memcpy(&block, in_b + block_idx, block_size - overflow_len);
			}

			uint32_t ta = (((ROL(a, 3) & ~ROR(e, 7)) ^ (~ROL(d, 5) & ROR(e, 7))) ^ ROR(block, 3));
			uint32_t tb = (ROR(block, 11) & ROR(a, 13)) ^ block;

			// compute something with the nums
			h = g;
			g = f;
			f = e;
			e = d + ta;
			d = c;
			c = b;
			b = a;
			a = ta + tb;
		}

		h0 = a;
		h1 = b;
		h2 = c;
		h3 = d;
		h4 = e;
		h5 = f;
		h6 = g;
		h7 = h;
	}

	// write result
	for (size_t i = 0; i < 8; ++i) {

		uint32_t num = h0;
		switch (i) {
		case 1:
			num = h1;
			break;
		case 2:
			num = h2;
			break;
		case 3:
			num = h3;
			break;
		case 4:
			num = h4;
			break;
		case 5:
			num = h5;
			break;
		case 6:
			num = h6;
			break;
		case 7:
			num = h7;
			break;
		}

		for (size_t j = 0; j < 4; ++j) {
			uint8_t res = ((num & (255ull << 24)) >> 24) % 256;
			size_t byte_idx = i * 4 + j;
			out_b[byte_idx] = res;
			num <<= 8;
		}
	}
}

__device__ void print_arr(const uint8_t * arr, size_t arr_len, size_t line_len) {
	for (size_t i = 0; i < arr_len; ++i) {
		printf("%d ", arr[i]);
		if ((i + 1) % line_len == 0)
			printf("\n");
	}
	printf("\n");
}

__global__ void kernel_hash(
	uint8_t *blocks,
	size_t blocks_pitch,
	uint32_t block_len,
	uint32_t nonce_idx,
	uint32_t nonce_len,
	bool *nonce_res,
	bool *done,
	uint8_t *thread_counters,
	size_t thread_counters_pitch,
	uint8_t *thread_hashes,
	size_t thread_hashes_pitch,
	size_t hash_len,
	size_t thread_count,
	uint32_t diff,
	size_t step_count) {
	
	// initialize global variable done
	size_t blockNumInGrid = blockIdx.x + gridDim.x  * blockIdx.y;
	size_t threadsPerBlock = blockDim.x * blockDim.y;
	size_t threadNumInBlock = threadIdx.x + blockDim.x * threadIdx.y;

	size_t globalThreadNum = blockNumInGrid * threadsPerBlock + threadNumInBlock;

	size_t thread_idx = globalThreadNum;
	size_t block_idx = thread_idx * (block_len + blocks_pitch);
	size_t thread_counter_idx = thread_idx * (thread_counters_pitch + nonce_len);
	size_t thread_hashes_idx = thread_idx * (thread_hashes_pitch + hash_len);
	bool overflow = false;

	*done = false;

	// clear the counter
	for (size_t i = 0; i < nonce_len; ++i) {
		thread_counters[thread_counter_idx + i] = 0;
	}

	// set the hash
	for (size_t i = 0; i < hash_len; ++i) {
		thread_hashes[thread_hashes_idx + i] = 255;
	}

	while (!overflow && !(*done)) {

		hash_256(blocks + block_idx, block_len, thread_hashes + thread_hashes_idx);
		//printf("%llu\n", size_t(blocks));
		//__syncthreads();

		uint32_t z_cnt = zero_count(thread_hashes + thread_hashes_idx, hash_len);
		if (z_cnt >= diff) {
			*done = true;
			nonce_res[thread_idx] = true;
		}
		else {
			// increment nonce
			for (size_t i = 0; !overflow && i < thread_count; ++i)
				overflow = increment(blocks + block_idx + nonce_idx, nonce_len);
			if (overflow) {
				*done = true;
			}

			// increment counter
			overflow = increment(thread_counters + thread_counter_idx, nonce_len);
			if (overflow) {
				*done = true;
			}
		}
	}
}

uint8_t* build_random_char_sequence(uint32_t len) {
	uint8_t *data = new uint8_t[len];
	for (uint32_t i = 0; i < len; ++i) {
		data[i] = rand() % 256;
	}
	return data;
}

void test_hash_256(size_t run_cnt) {

	uint8_t hash[32]{ 0 };
	uint8_t *bytes = build_random_char_sequence(15);
	std::memset(bytes + 3, 0, 5);
	
	while (run_cnt--) {
		hash_256(bytes, 15, hash);
		increment(bytes + 3, 5);
	}
}

int main() {
	// host data
	const size_t step_count = 100;
	const uint32_t block_count = 1;
	const uint32_t threads_per_block = 500;
	const uint32_t thread_count = block_count * threads_per_block;
	const uint32_t block_len = 26;
	const uint32_t nonce_idx = 5;
	const uint32_t nonce_len = 10;
	const uint32_t diff = 16;
	const uint8_t hash_len = 32;

	bool *nonce_res;
	uint8_t *blocks;
	int64_t correct_idx = -1;
	uint8_t counter[nonce_len];
	uint8_t *block = build_random_char_sequence(block_len);

	// result nonce
	uint8_t nonce[nonce_len]{ 0 };
	uint8_t *counters;

	// device data
	uint8_t *d_blocks;
	bool *d_nonce_res;
	bool *d_done;
	uint8_t *d_thread_counters;
	uint8_t *d_thread_hashes;

	// alloc space
	nonce_res = new bool[thread_count];
	for (size_t i = 0; i < thread_count; ++i)
		nonce_res[i] = false;

	CUDA_HANDLE_ERR(hipMalloc(&d_nonce_res, thread_count * sizeof(bool)), "Error on malloc for d_nonce_res");
	
	blocks = new uint8_t[block_len * thread_count];
	CUDA_HANDLE_ERR(hipMalloc(&d_blocks, thread_count * block_len), "Error on malloc for d_blocks");
	
	CUDA_HANDLE_ERR(hipMalloc(&d_done, sizeof(bool)), "Error on malloc for d_done");
	
	counters = new uint8_t[thread_count * nonce_len];
	CUDA_HANDLE_ERR(hipMalloc(&d_thread_counters, thread_count * nonce_len), "Error on malloc for d_thread_counters");
	
	CUDA_HANDLE_ERR(hipMalloc(&d_thread_hashes, thread_count * hash_len), "Error on malloc for d_thread_hashes");

	for (size_t i = 0; i < thread_count; ++i)
		nonce_res[i] = false;

	std::memset(counter, 0, nonce_len);
	// prep block data
	for (size_t i = 0; i < thread_count; ++i) {
		if (i)
			increment(counter, nonce_len);
	
		std::memcpy(blocks + (i * block_len), block, block_len);
		std::memcpy(blocks + (i * block_len) + nonce_idx, counter, nonce_len);
	}

	CUDA_HANDLE_ERR(hipMemcpy(d_blocks, blocks, thread_count * block_len, hipMemcpyHostToDevice), "Error on memcpy for blocks to d_blocks");
	CUDA_HANDLE_ERR(hipMemcpy(d_nonce_res, nonce_res, thread_count * sizeof(bool), hipMemcpyHostToDevice), "Error on memcpy for nonce_res to d_nonce_res");

	bool error_occured = false;

	do {
		kernel_hash<<<block_count, threads_per_block>>>(
			d_blocks, 0, block_len, nonce_idx, nonce_len, 
			d_nonce_res, 
			d_done, 
			d_thread_counters, 0, 
			d_thread_hashes, 0, hash_len, 
			thread_count, diff,
			step_count);

		CUDA_HANDLE_ERR_BOOL_FLAG(hipGetLastError(), "Sync kernel error", error_occured);
		CUDA_HANDLE_ERR_BOOL_FLAG(hipDeviceSynchronize(), "Async kernel error", error_occured);
		CUDA_HANDLE_ERR_BOOL_FLAG(hipMemcpy(nonce_res, d_nonce_res, thread_count * sizeof(bool), hipMemcpyDeviceToHost), "Error on getting nonce_res for data", error_occured);
		CUDA_HANDLE_ERR_BOOL_FLAG(hipMemcpy(counters, d_thread_counters, thread_count * nonce_len, hipMemcpyDeviceToHost), "Error on getting counters from device", error_occured);
		
		for (uint32_t i = 0; correct_idx == -1 && i < thread_count; ++i)
			if (nonce_res[i])
				correct_idx = i;

	} while (!error_occured && correct_idx == -1);

	if (correct_idx == -1)
		goto cleanup;

	// do something with the nonce
	std::cout << correct_idx << "\n";
	for (uint8_t step_counter[nonce_len]{ 0 }; memcmp(step_counter, counters + correct_idx * nonce_len, nonce_len) != 0; increment(step_counter, nonce_len)) {
		for (size_t i = 0; i < thread_count; ++i)
			increment(nonce, nonce_len);
	}

	// find the starting nonce of the 'winning' thread
	while (correct_idx--)
		increment(nonce, nonce_len);

	// print the nonce
	for (size_t i = 0; i < nonce_len; ++i)
		std::cout << int(nonce[i]) << " ";
	std::cout << "\n";

	// print the hash
	for (size_t i = 0; i < nonce_len; ++i)
		block[nonce_idx + i] = nonce[i];

	uint8_t hash_res[32]{ 0 };
	hash_256(block, block_len, hash_res);
	
	for (size_t i = 0; i < hash_len; ++i)
		std::cout << int(hash_res[i]) << " ";
	std::cout << "\n";

cleanup:

	hipFree(d_nonce_res);
	hipFree(d_blocks);
	hipFree(d_done);
	hipFree(d_thread_counters);
	hipFree(d_thread_hashes);

	delete[] counters;
	delete[] nonce_res;
	delete[] blocks;
	delete[] block;
	return 0;
}
